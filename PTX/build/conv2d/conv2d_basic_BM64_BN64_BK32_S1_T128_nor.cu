#include "hip/hip_runtime.h"
#include <tl_templates/cuda/gemm.h>
#include <tl_templates/cuda/copy.h>
#include <tl_templates/cuda/reduce.h>
#include <tl_templates/cuda/ldsm.h>
#include <tl_templates/cuda/threadblock_swizzle.h>
#include <tl_templates/cuda/debug.h>
#ifdef ENABLE_BF16
#include <tl_templates/cuda/cuda_bf16_fallbacks.cuh>
#endif

extern "C" __global__ void main_kernel(half_t* __restrict__ data, half_t* __restrict__ out_flat, half_t* __restrict__ w_flat);
extern "C" __global__ void __launch_bounds__(128, 1) main_kernel(half_t* __restrict__ data, half_t* __restrict__ out_flat, half_t* __restrict__ w_flat) {
  extern __shared__ __align__(1024) uchar buf_dyn_shmem[];
  float out_local[32];
  #pragma unroll
  for (int i = 0; i < 8; ++i) {
    *(float4*)(out_local + (i * 4)) = make_float4(0x0p+0f/*0.000000e+00*/, 0x0p+0f/*0.000000e+00*/, 0x0p+0f/*0.000000e+00*/, 0x0p+0f/*0.000000e+00*/);
  }
  for (int kt = 0; kt < 36; ++kt) {
    #pragma unroll
    for (int i_1 = 0; i_1 < 2; ++i_1) {
      bool inb = ((((1 <= ((kt / 12) + (((int)blockIdx.y) & 63))) && (((kt / 12) + (((int)blockIdx.y) & 63)) < 65)) && (1 <= (((i_1 * 32) + (((int)threadIdx.x) >> 2)) + ((kt % 12) >> 2)))) && ((((i_1 * 32) + (((int)threadIdx.x) >> 2)) + ((kt % 12) >> 2)) < 65));
      uint4 condval;
      if ((((((1 <= (((i_1 * 32) + (((int)threadIdx.x) >> 2)) + ((kt % 12) >> 2))) && ((((i_1 * 32) + (((int)threadIdx.x) >> 2)) + ((kt % 12) >> 2)) < 65)) && (1 <= ((kt / 12) + (((int)blockIdx.y) & 63)))) && (((kt / 12) + (((int)blockIdx.y) & 63)) < 65)) && inb)) {
        condval = *(uint4*)(data + ((((((((kt / 12) * 8192) + (((int)blockIdx.y) * 8192)) + (i_1 * 4096)) + ((((int)threadIdx.x) >> 2) * 128)) + ((kt % 12) * 32)) + ((((int)threadIdx.x) & 3) * 8)) - 8320));
      } else {
        condval = make_uint4(__pack_half2(half_t(0x0p+0f/*0.000000e+00*/), half_t(0x0p+0f/*0.000000e+00*/)), __pack_half2(half_t(0x0p+0f/*0.000000e+00*/), half_t(0x0p+0f/*0.000000e+00*/)), __pack_half2(half_t(0x0p+0f/*0.000000e+00*/), half_t(0x0p+0f/*0.000000e+00*/)), __pack_half2(half_t(0x0p+0f/*0.000000e+00*/), half_t(0x0p+0f/*0.000000e+00*/)));
      }
      *(uint4*)(((half_t*)buf_dyn_shmem) + (((i_1 * 1024) + (((int)threadIdx.x) * 8)) + 2048)) = condval;
    }
    #pragma unroll
    for (int i_2 = 0; i_2 < 2; ++i_2) {
      *(uint4*)(((half_t*)buf_dyn_shmem) + ((i_2 * 1024) + (((int)threadIdx.x) * 8))) = *(uint4*)(w_flat + (((((kt * 4096) + (i_2 * 2048)) + ((((int)threadIdx.x) >> 3) * 128)) + (((int)blockIdx.x) * 64)) + ((((int)threadIdx.x) & 7) * 8)));
    }
    __syncthreads();
    for (int kk = 0; kk < 32; ++kk) {
      #pragma unroll
      for (int i_3 = 0; i_3 < 32; ++i_3) {
        out_local[i_3] = (out_local[i_3] + (((float)((half_t*)buf_dyn_shmem)[(((((i_3 >> 2) * 256) + ((((int)threadIdx.x) >> 4) * 32)) + kk) + 2048)]) * ((float)((half_t*)buf_dyn_shmem)[(((kk * 64) + ((((int)threadIdx.x) & 15) * 4)) + (i_3 & 3))])));
      }
    }
    __syncthreads();
  }
  #pragma unroll
  for (int i_4 = 0; i_4 < 8; ++i_4) {
    uint2 __1;
    float4 v_ = *(float4*)(out_local + (i_4 * 4));
    ((half2*)(&(__1.x)))->x = (half_t)(v_.x);
    ((half2*)(&(__1.x)))->y = (half_t)(v_.y);
    ((half2*)(&(__1.y)))->x = (half_t)(v_.z);
    ((half2*)(&(__1.y)))->y = (half_t)(v_.w);
    *(uint2*)(((half_t*)buf_dyn_shmem) + ((i_4 * 512) + (((int)threadIdx.x) * 4))) = __1;
  }
  __syncthreads();
  #pragma unroll
  for (int i_5 = 0; i_5 < 4; ++i_5) {
    *(uint4*)(out_flat + (((((((int)blockIdx.y) * 8192) + (i_5 * 2048)) + ((((int)threadIdx.x) >> 3) * 128)) + (((int)blockIdx.x) * 64)) + ((((int)threadIdx.x) & 7) * 8))) = *(uint4*)(((half_t*)buf_dyn_shmem) + ((i_5 * 1024) + (((int)threadIdx.x) * 8)));
  }
}


#define ERROR_BUF_SIZE 1024
static char error_buf[ERROR_BUF_SIZE];

extern "C" const char* get_last_error() {
    return error_buf;
}

extern "C" int init() {
    error_buf[0] = '\0';
    
    hipError_t result_main_kernel = hipFuncSetAttribute(reinterpret_cast<const void*>(main_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, 8192);
    if (result_main_kernel != hipSuccess) {
        snprintf(error_buf, ERROR_BUF_SIZE, "Failed to set the allowed dynamic shared memory size to %d with error: %s", 8192, hipGetErrorString(result_main_kernel));
        return -1;
    }

    return 0;
}

extern "C" int call(half_t* __restrict__ data, half_t* __restrict__ w_flat, half_t* __restrict__ out_flat, hipStream_t stream=hipStreamDefault) {
	main_kernel<<<dim3(2, 8192, 1), dim3(128, 1, 1), 8192, stream>>>(data, out_flat, w_flat);
	TILELANG_CHECK_LAST_ERROR("main_kernel");

	return 0;
}
