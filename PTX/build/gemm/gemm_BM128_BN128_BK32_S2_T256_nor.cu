#include "hip/hip_runtime.h"
#include <tl_templates/cuda/gemm.h>
#include <tl_templates/cuda/copy.h>
#include <tl_templates/cuda/reduce.h>
#include <tl_templates/cuda/ldsm.h>
#include <tl_templates/cuda/threadblock_swizzle.h>
#include <tl_templates/cuda/debug.h>

extern "C" __global__ void main_kernel(half_t* __restrict__ A, half_t* __restrict__ B, half_t* __restrict__ C);
extern "C" __global__ void __launch_bounds__(256, 1) main_kernel(half_t* __restrict__ A, half_t* __restrict__ B, half_t* __restrict__ C) {
  extern __shared__ __align__(1024) uchar buf_dyn_shmem[];
  float C_local[64];
  #pragma unroll
  for (int i = 0; i < 32; ++i) {
    *(float2*)(C_local + (i * 2)) = make_float2(0.000000e+00f, 0.000000e+00f);
  }
  #pragma unroll
  for (int i_1 = 0; i_1 < 2; ++i_1) {
    tl::cp_async_gs<16>(buf_dyn_shmem+((((i_1 * 4096) + ((((int)threadIdx.x) >> 2) * 64)) + (((((((int)threadIdx.x) & 31) >> 4) + ((((int)threadIdx.x) & 3) >> 1)) & 1) * 32)) + (((((((int)threadIdx.x) & 15) >> 3) + (((int)threadIdx.x) & 1)) & 1) * 16)), A+((((((int)blockIdx.y) * 524288) + (i_1 * 262144)) + ((((int)threadIdx.x) >> 2) * 4096)) + ((((int)threadIdx.x) & 3) * 8)));
  }
  #pragma unroll
  for (int i_2 = 0; i_2 < 2; ++i_2) {
    tl::cp_async_gs<16>(buf_dyn_shmem+(((((i_2 * 4096) + ((((int)threadIdx.x) >> 2) * 64)) + (((((((int)threadIdx.x) & 31) >> 4) + ((((int)threadIdx.x) & 3) >> 1)) & 1) * 32)) + (((((((int)threadIdx.x) & 15) >> 3) + (((int)threadIdx.x) & 1)) & 1) * 16)) + 16384), B+((((((int)blockIdx.x) * 524288) + (i_2 * 262144)) + ((((int)threadIdx.x) >> 2) * 4096)) + ((((int)threadIdx.x) & 3) * 8)));
  }
  tl::cp_async_commit();
  for (int k = 0; k < 127; ++k) {
    __syncthreads();
    #pragma unroll
    for (int i_3 = 0; i_3 < 2; ++i_3) {
      tl::cp_async_gs<16>(buf_dyn_shmem+(((((((k + 1) & 1) * 8192) + (i_3 * 4096)) + ((((int)threadIdx.x) >> 2) * 64)) + (((((((int)threadIdx.x) & 31) >> 4) + ((((int)threadIdx.x) & 3) >> 1)) & 1) * 32)) + (((((((int)threadIdx.x) & 15) >> 3) + (((int)threadIdx.x) & 1)) & 1) * 16)), A+((((((((int)blockIdx.y) * 524288) + (i_3 * 262144)) + ((((int)threadIdx.x) >> 2) * 4096)) + (k * 32)) + ((((int)threadIdx.x) & 3) * 8)) + 32));
    }
    #pragma unroll
    for (int i_4 = 0; i_4 < 2; ++i_4) {
      tl::cp_async_gs<16>(buf_dyn_shmem+((((((((k + 1) & 1) * 8192) + (i_4 * 4096)) + ((((int)threadIdx.x) >> 2) * 64)) + (((((((int)threadIdx.x) & 31) >> 4) + ((((int)threadIdx.x) & 3) >> 1)) & 1) * 32)) + (((((((int)threadIdx.x) & 15) >> 3) + (((int)threadIdx.x) & 1)) & 1) * 16)) + 16384), B+((((((((int)blockIdx.x) * 524288) + (i_4 * 262144)) + ((((int)threadIdx.x) >> 2) * 4096)) + (k * 32)) + ((((int)threadIdx.x) & 3) * 8)) + 32));
    }
    tl::cp_async_commit();
    tl::cp_async_wait<1>();
    __syncthreads();
    tl::gemm_ss<128, 128, 32, 2, 4, 0, 1, 0>((&(((half_t*)buf_dyn_shmem)[((k & 1) * 4096)])), (&(((half_t*)buf_dyn_shmem)[(((k & 1) * 4096) + 8192)])), (&(C_local[0])));
  }
  tl::cp_async_wait<0>();
  __syncthreads();
  tl::gemm_ss<128, 128, 32, 2, 4, 0, 1, 0>((&(((half_t*)buf_dyn_shmem)[4096])), (&(((half_t*)buf_dyn_shmem)[12288])), (&(C_local[0])));
  #pragma unroll
  for (int i_5 = 0; i_5 < 32; ++i_5) {
    uint1 __1;
    float2 v_ = *(float2*)(C_local + (i_5 * 2));
    ((half2*)(&(__1.x)))->x = (half_t)(v_.x);
    ((half2*)(&(__1.x)))->y = (half_t)(v_.y);
    *(uint1*)(C + (((((((((((int)blockIdx.y) * 524288) + (((i_5 & 7) >> 1) * 131072)) + (((((int)threadIdx.x) & 63) >> 5) * 65536)) + ((i_5 & 1) * 32768)) + (((((int)threadIdx.x) & 31) >> 2) * 4096)) + (((int)blockIdx.x) * 128)) + ((i_5 >> 3) * 32)) + ((((int)threadIdx.x) >> 6) * 8)) + ((((int)threadIdx.x) & 3) * 2))) = __1;
  }
}


#define ERROR_BUF_SIZE 1024
static char error_buf[ERROR_BUF_SIZE];

extern "C" const char* get_last_error() {
    return error_buf;
}

extern "C" int init() {
    error_buf[0] = '\0';
    
    hipError_t result_main_kernel = hipFuncSetAttribute(reinterpret_cast<const void*>(main_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, 32768);
    if (result_main_kernel != hipSuccess) {
        snprintf(error_buf, ERROR_BUF_SIZE, "Failed to set the allowed dynamic shared memory size to %d with error: %s", 32768, hipGetErrorString(result_main_kernel));
        return -1;
    }

    return 0;
}

extern "C" int call(half_t* __restrict__ A, half_t* __restrict__ B, half_t* __restrict__ C, hipStream_t stream=hipStreamDefault) {
	main_kernel<<<dim3(32, 32, 1), dim3(256, 1, 1), 32768, stream>>>(A, B, C);
	TILELANG_CHECK_LAST_ERROR("main_kernel");

	return 0;
}
