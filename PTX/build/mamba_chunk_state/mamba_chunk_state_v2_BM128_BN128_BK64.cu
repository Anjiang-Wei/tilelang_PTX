#include "hip/hip_runtime.h"
#include <tl_templates/cuda/gemm.h>
#include <tl_templates/cuda/copy.h>
#include <tl_templates/cuda/reduce.h>
#include <tl_templates/cuda/ldsm.h>
#include <tl_templates/cuda/threadblock_swizzle.h>
#include <tl_templates/cuda/debug.h>

extern "C" __global__ void main_kernel(half_t* __restrict__ A, half_t* __restrict__ B, half_t* __restrict__ C);
extern "C" __global__ void __launch_bounds__(256, 1) main_kernel(half_t* __restrict__ A, half_t* __restrict__ B, half_t* __restrict__ C) {
  extern __shared__ __align__(1024) uchar buf_dyn_shmem[];
  float C_local[64];
  #pragma unroll
  for (int i = 0; i < 32; ++i) {
    *(float2*)(C_local + (i * 2)) = make_float2(0.000000e+00f, 0.000000e+00f);
  }
  for (int k = 0; k < 16; ++k) {
    __syncthreads();
    #pragma unroll
    for (int i_1 = 0; i_1 < 4; ++i_1) {
      *(uint4*)(((half_t*)buf_dyn_shmem) + (((((i_1 * 2048) + ((((int)threadIdx.x) >> 3) * 64)) + (((((((int)threadIdx.x) & 63) >> 5) + ((((int)threadIdx.x) & 7) >> 2)) & 1) * 32)) + (((((((int)threadIdx.x) & 31) >> 4) + ((((int)threadIdx.x) & 3) >> 1)) & 1) * 16)) + (((((((int)threadIdx.x) & 15) >> 3) + (((int)threadIdx.x) & 1)) & 1) * 8))) = *(uint4*)(A + (((((((int)blockIdx.y) * 131072) + (i_1 * 32768)) + ((((int)threadIdx.x) >> 3) * 1024)) + (k * 64)) + ((((int)threadIdx.x) & 7) * 8)));
    }
    #pragma unroll
    for (int i_2 = 0; i_2 < 4; ++i_2) {
      *(uint4*)(((half_t*)buf_dyn_shmem) + ((((((i_2 * 2048) + ((((int)threadIdx.x) >> 3) * 64)) + (((((((int)threadIdx.x) & 63) >> 5) + ((((int)threadIdx.x) & 7) >> 2)) & 1) * 32)) + (((((((int)threadIdx.x) & 31) >> 4) + ((((int)threadIdx.x) & 3) >> 1)) & 1) * 16)) + (((((((int)threadIdx.x) & 15) >> 3) + (((int)threadIdx.x) & 1)) & 1) * 8)) + 8192)) = *(uint4*)(B + (((((((int)blockIdx.x) * 131072) + (i_2 * 32768)) + ((((int)threadIdx.x) >> 3) * 1024)) + (k * 64)) + ((((int)threadIdx.x) & 7) * 8)));
    }
    __syncthreads();
    tl::gemm_ss<128, 128, 64, 2, 4, 0, 1, 0>((&(((half_t*)buf_dyn_shmem)[0])), (&(((half_t*)buf_dyn_shmem)[8192])), (&(C_local[0])));
  }
  #pragma unroll
  for (int i_3 = 0; i_3 < 32; ++i_3) {
    uint1 __1;
    float2 v_ = *(float2*)(C_local + (i_3 * 2));
    ((half2*)(&(__1.x)))->x = (half_t)(v_.x);
    ((half2*)(&(__1.x)))->y = (half_t)(v_.y);
    *(uint1*)(C + (((((((((((int)blockIdx.y) * 131072) + (((i_3 & 7) >> 1) * 32768)) + (((((int)threadIdx.x) & 63) >> 5) * 16384)) + ((i_3 & 1) * 8192)) + (((((int)threadIdx.x) & 31) >> 2) * 1024)) + (((int)blockIdx.x) * 128)) + ((i_3 >> 3) * 32)) + ((((int)threadIdx.x) >> 6) * 8)) + ((((int)threadIdx.x) & 3) * 2))) = __1;
  }
}


#define ERROR_BUF_SIZE 1024
static char error_buf[ERROR_BUF_SIZE];

extern "C" const char* get_last_error() {
    return error_buf;
}

extern "C" int init() {
    error_buf[0] = '\0';
    
    hipError_t result_main_kernel = hipFuncSetAttribute(reinterpret_cast<const void*>(main_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, 32768);
    if (result_main_kernel != hipSuccess) {
        snprintf(error_buf, ERROR_BUF_SIZE, "Failed to set the allowed dynamic shared memory size to %d with error: %s", 32768, hipGetErrorString(result_main_kernel));
        return -1;
    }

    return 0;
}

extern "C" int call(half_t* __restrict__ A, half_t* __restrict__ B, half_t* __restrict__ C, hipStream_t stream=hipStreamDefault) {
	main_kernel<<<dim3(8, 8, 1), dim3(256, 1, 1), 32768, stream>>>(A, B, C);
	TILELANG_CHECK_LAST_ERROR("main_kernel");

	return 0;
}
