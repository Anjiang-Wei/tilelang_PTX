#include "hip/hip_runtime.h"
#include <tl_templates/cuda/gemm.h>
#include <tl_templates/cuda/copy.h>
#include <tl_templates/cuda/reduce.h>
#include <tl_templates/cuda/ldsm.h>
#include <tl_templates/cuda/threadblock_swizzle.h>
#include <tl_templates/cuda/debug.h>
#ifdef ENABLE_BF16
#include <tl_templates/cuda/cuda_bf16_fallbacks.cuh>
#endif

extern "C" __global__ void main_kernel(half_t* __restrict__ A, half_t* __restrict__ B, half_t* __restrict__ C);
extern "C" __global__ void __launch_bounds__(512, 1) main_kernel(half_t* __restrict__ A, half_t* __restrict__ B, half_t* __restrict__ C) {
  extern __shared__ __align__(1024) uchar buf_dyn_shmem[];
  float C_loc[8];
  #pragma unroll
  for (int i = 0; i < 4; ++i) {
    *(float2*)(C_loc + (i * 2)) = make_float2(0x0p+0f/*0.000000e+00*/, 0x0p+0f/*0.000000e+00*/);
  }
  for (int kt = 0; kt < 128; ++kt) {
    *(uint2*)(((half_t*)buf_dyn_shmem) + (((((((int)threadIdx.x) >> 3) * 32) + (((((((int)threadIdx.x) & 63) >> 5) + ((((int)threadIdx.x) & 7) >> 2)) & 1) * 16)) + (((((((int)threadIdx.x) & 31) >> 4) + ((((int)threadIdx.x) & 3) >> 1)) & 1) * 8)) + ((((int)threadIdx.x) & 1) * 4))) = *(uint2*)(A + (((((((int)blockIdx.x) >> 6) * 262144) + ((((int)threadIdx.x) >> 3) * 4096)) + (kt * 32)) + ((((int)threadIdx.x) & 7) * 4)));
    *(uint2*)(((half_t*)buf_dyn_shmem) + ((((((((int)threadIdx.x) >> 3) * 32) + (((((((int)threadIdx.x) & 63) >> 5) + ((((int)threadIdx.x) & 7) >> 2)) & 1) * 16)) + (((((((int)threadIdx.x) & 31) >> 4) + ((((int)threadIdx.x) & 3) >> 1)) & 1) * 8)) + ((((int)threadIdx.x) & 1) * 4)) + 2048)) = *(uint2*)(B + (((((((int)blockIdx.x) & 63) * 262144) + ((((int)threadIdx.x) >> 3) * 4096)) + (kt * 32)) + ((((int)threadIdx.x) & 7) * 4)));
    __syncthreads();
    tl::gemm_ss<64, 64, 32, 4, 4, 0, 1, 0, 32, 32, 0, 0>((&(((half_t*)buf_dyn_shmem)[0])), (&(((half_t*)buf_dyn_shmem)[2048])), (&(C_loc[0])));
    __syncthreads();
  }
  #pragma unroll
  for (int i_1 = 0; i_1 < 4; ++i_1) {
    uint1 __1;
    float2 v_ = *(float2*)(C_loc + (i_1 * 2));
    ((half2*)(&(__1.x)))->x = (half_t)(v_.x);
    ((half2*)(&(__1.x)))->y = (half_t)(v_.y);
    *(uint1*)(C + (((((((((((int)blockIdx.x) >> 6) * 262144) + (((((int)threadIdx.x) & 127) >> 5) * 65536)) + ((i_1 & 1) * 32768)) + (((((int)threadIdx.x) & 31) >> 2) * 4096)) + ((((int)blockIdx.x) & 63) * 64)) + ((i_1 >> 1) * 32)) + ((((int)threadIdx.x) >> 7) * 8)) + ((((int)threadIdx.x) & 3) * 2))) = __1;
  }
}


#define ERROR_BUF_SIZE 1024
static char error_buf[ERROR_BUF_SIZE];

extern "C" const char* get_last_error() {
    return error_buf;
}

extern "C" int init() {
    error_buf[0] = '\0';
    
    hipError_t result_main_kernel = hipFuncSetAttribute(reinterpret_cast<const void*>(main_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, 8192);
    if (result_main_kernel != hipSuccess) {
        snprintf(error_buf, ERROR_BUF_SIZE, "Failed to set the allowed dynamic shared memory size to %d with error: %s", 8192, hipGetErrorString(result_main_kernel));
        return -1;
    }

    return 0;
}

extern "C" int call(half_t* __restrict__ A, half_t* __restrict__ B, half_t* __restrict__ C, hipStream_t stream=hipStreamDefault) {
	main_kernel<<<dim3(4096, 1, 1), dim3(512, 1, 1), 8192, stream>>>(A, B, C);
	TILELANG_CHECK_LAST_ERROR("main_kernel");

	return 0;
}
