#include "hip/hip_runtime.h"
#include <tl_templates/cuda/gemm.h>
#include <tl_templates/cuda/copy.h>
#include <tl_templates/cuda/reduce.h>
#include <tl_templates/cuda/ldsm.h>
#include <tl_templates/cuda/threadblock_swizzle.h>
#include <tl_templates/cuda/debug.h>
#ifdef ENABLE_BF16
#include <tl_templates/cuda/cuda_bf16_fallbacks.cuh>
#endif

extern "C" __global__ void main_kernel(half_t* __restrict__ A, half_t* __restrict__ B, half_t* __restrict__ C);
extern "C" __global__ void __launch_bounds__(256, 1) main_kernel(half_t* __restrict__ A, half_t* __restrict__ B, half_t* __restrict__ C) {
  extern __shared__ __align__(1024) uchar buf_dyn_shmem[];
  float C_local[32];
  #pragma unroll
  for (int i = 0; i < 16; ++i) {
    *(float2*)(C_local + (i * 2)) = make_float2(0x0p+0f/*0.000000e+00*/, 0x0p+0f/*0.000000e+00*/);
  }
  for (int k = 0; k < 128; ++k) {
    __syncthreads();
    #pragma unroll
    for (int i_1 = 0; i_1 < 2; ++i_1) {
      *(uint4*)(((half_t*)buf_dyn_shmem) + ((((i_1 * 2048) + ((((int)threadIdx.x) >> 2) * 32)) + (((((((int)threadIdx.x) & 31) >> 4) + ((((int)threadIdx.x) & 3) >> 1)) & 1) * 16)) + (((((((int)threadIdx.x) & 15) >> 3) + (((int)threadIdx.x) & 1)) & 1) * 8))) = *(uint4*)(A + (((((((int)blockIdx.y) * 524288) + (i_1 * 262144)) + ((((int)threadIdx.x) >> 2) * 4096)) + (k * 32)) + ((((int)threadIdx.x) & 3) * 8)));
    }
    *(uint4*)(((half_t*)buf_dyn_shmem) + (((((((int)threadIdx.x) >> 2) * 32) + (((((((int)threadIdx.x) & 31) >> 4) + ((((int)threadIdx.x) & 3) >> 1)) & 1) * 16)) + (((((((int)threadIdx.x) & 15) >> 3) + (((int)threadIdx.x) & 1)) & 1) * 8)) + 4096)) = *(uint4*)(B + ((((((int)blockIdx.x) * 262144) + ((((int)threadIdx.x) >> 2) * 4096)) + (k * 32)) + ((((int)threadIdx.x) & 3) * 8)));
    __syncthreads();
    tl::gemm_ss<128, 64, 32, 2, 4, 0, 1, 0, 32, 32, 0, 0>((&(((half_t*)buf_dyn_shmem)[0])), (&(((half_t*)buf_dyn_shmem)[4096])), (&(C_local[0])));
  }
  #pragma unroll
  for (int i_2 = 0; i_2 < 16; ++i_2) {
    uint1 __1;
    float2 v_ = *(float2*)(C_local + (i_2 * 2));
    ((half2*)(&(__1.x)))->x = (half_t)(v_.x);
    ((half2*)(&(__1.x)))->y = (half_t)(v_.y);
    *(uint1*)(C + (((((((((((int)blockIdx.y) * 524288) + (((i_2 & 7) >> 1) * 131072)) + (((((int)threadIdx.x) & 63) >> 5) * 65536)) + ((i_2 & 1) * 32768)) + (((((int)threadIdx.x) & 31) >> 2) * 4096)) + (((int)blockIdx.x) * 64)) + ((i_2 >> 3) * 32)) + ((((int)threadIdx.x) >> 6) * 8)) + ((((int)threadIdx.x) & 3) * 2))) = __1;
  }
}


#define ERROR_BUF_SIZE 1024
static char error_buf[ERROR_BUF_SIZE];

extern "C" const char* get_last_error() {
    return error_buf;
}

extern "C" int init() {
    error_buf[0] = '\0';
    
    hipError_t result_main_kernel = hipFuncSetAttribute(reinterpret_cast<const void*>(main_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, 12288);
    if (result_main_kernel != hipSuccess) {
        snprintf(error_buf, ERROR_BUF_SIZE, "Failed to set the allowed dynamic shared memory size to %d with error: %s", 12288, hipGetErrorString(result_main_kernel));
        return -1;
    }

    return 0;
}

extern "C" int call(half_t* __restrict__ A, half_t* __restrict__ B, half_t* __restrict__ C, hipStream_t stream=hipStreamDefault) {
	main_kernel<<<dim3(64, 32, 1), dim3(256, 1, 1), 12288, stream>>>(A, B, C);
	TILELANG_CHECK_LAST_ERROR("main_kernel");

	return 0;
}
