#include "hip/hip_runtime.h"
#include <tl_templates/cuda/gemm.h>
#include <tl_templates/cuda/copy.h>
#include <tl_templates/cuda/reduce.h>
#include <tl_templates/cuda/ldsm.h>
#include <tl_templates/cuda/threadblock_swizzle.h>
#include <tl_templates/cuda/debug.h>

extern "C" __global__ void chunk_linear_attn_fwd_kernel(half_t* __restrict__ K, half_t* __restrict__ O, half_t* __restrict__ Q, half_t* __restrict__ V, float* __restrict__ final_state);
extern "C" __global__ void __launch_bounds__(128, 1) chunk_linear_attn_fwd_kernel(half_t* __restrict__ K, half_t* __restrict__ O, half_t* __restrict__ Q, half_t* __restrict__ V, float* __restrict__ final_state) {
  extern __shared__ __align__(1024) uchar buf_dyn_shmem[];
  float h[8];
  float s[32];
  float o[16];
  #pragma unroll
  for (int i = 0; i < 4; ++i) {
    *(float2*)(h + (i * 2)) = make_float2(0.000000e+00f, 0.000000e+00f);
  }
  const dim3 blockIdx = tl::rasterization2DRow<10>();
  #pragma unroll
  for (int i_1 = 0; i_1 < 2; ++i_1) {
    tl::cp_async_gs<16>(buf_dyn_shmem+(((((i_1 * 2048) + ((((int)threadIdx.x) >> 2) * 64)) + (((((((int)threadIdx.x) & 31) >> 4) + ((((int)threadIdx.x) & 3) >> 1)) & 1) * 32)) + (((((((int)threadIdx.x) & 15) >> 3) + (((int)threadIdx.x) & 1)) & 1) * 16)) + 10240), K+(((((((((int)blockIdx.z) >> 2) * 32768) + (i_1 * 8192)) + ((((int)threadIdx.x) >> 2) * 256)) + ((((int)blockIdx.z) & 3) * 64)) + (((int)blockIdx.y) * 32)) + ((((int)threadIdx.x) & 3) * 8)));
  }
  #pragma unroll
  for (int i_2 = 0; i_2 < 2; ++i_2) {
    tl::cp_async_gs<16>(buf_dyn_shmem+(((((i_2 * 2048) + ((((int)threadIdx.x) >> 2) * 64)) + (((((((int)threadIdx.x) & 31) >> 4) + ((((int)threadIdx.x) & 3) >> 1)) & 1) * 32)) + (((((((int)threadIdx.x) & 15) >> 3) + (((int)threadIdx.x) & 1)) & 1) * 16)) + 18432), V+(((((((((int)blockIdx.z) >> 2) * 32768) + (i_2 * 8192)) + ((((int)threadIdx.x) >> 2) * 256)) + ((((int)blockIdx.z) & 3) * 64)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 3) * 8)));
  }
  tl::cp_async_commit();
  #pragma unroll
  for (int i_3 = 0; i_3 < 2; ++i_3) {
    uint4 __1;
    ulonglong4 __2;
      ulonglong4 __3;
      uint4 v_ = *(uint4*)(Q + (((((((((int)blockIdx.z) >> 2) * 32768) + (i_3 * 8192)) + ((((int)threadIdx.x) >> 2) * 256)) + ((((int)blockIdx.z) & 3) * 64)) + (((int)blockIdx.y) * 32)) + ((((int)threadIdx.x) & 3) * 8)));
      ((float2*)(&(__3.x)))->x = (float)(((half2*)(&(v_.x)))->x);
      ((float2*)(&(__3.x)))->y = (float)(((half2*)(&(v_.x)))->y);
      ((float2*)(&(__3.y)))->x = (float)(((half2*)(&(v_.y)))->x);
      ((float2*)(&(__3.y)))->y = (float)(((half2*)(&(v_.y)))->y);
      ((float2*)(&(__3.z)))->x = (float)(((half2*)(&(v_.z)))->x);
      ((float2*)(&(__3.z)))->y = (float)(((half2*)(&(v_.z)))->y);
      ((float2*)(&(__3.w)))->x = (float)(((half2*)(&(v_.w)))->x);
      ((float2*)(&(__3.w)))->y = (float)(((half2*)(&(v_.w)))->y);
      ulonglong4 v__1 = make_ulonglong4(*(unsigned long long*)&make_float2(1.250000e-01f, 1.250000e-01f), *(unsigned long long*)&make_float2(1.250000e-01f, 1.250000e-01f), *(unsigned long long*)&make_float2(1.250000e-01f, 1.250000e-01f), *(unsigned long long*)&make_float2(1.250000e-01f, 1.250000e-01f));
      ((float2*)(&(__2.x)))->x = (((float2*)(&(__3.x)))->x*((float2*)(&(v__1.x)))->x);
      ((float2*)(&(__2.x)))->y = (((float2*)(&(__3.x)))->y*((float2*)(&(v__1.x)))->y);
      ((float2*)(&(__2.y)))->x = (((float2*)(&(__3.y)))->x*((float2*)(&(v__1.y)))->x);
      ((float2*)(&(__2.y)))->y = (((float2*)(&(__3.y)))->y*((float2*)(&(v__1.y)))->y);
      ((float2*)(&(__2.z)))->x = (((float2*)(&(__3.z)))->x*((float2*)(&(v__1.z)))->x);
      ((float2*)(&(__2.z)))->y = (((float2*)(&(__3.z)))->y*((float2*)(&(v__1.z)))->y);
      ((float2*)(&(__2.w)))->x = (((float2*)(&(__3.w)))->x*((float2*)(&(v__1.w)))->x);
      ((float2*)(&(__2.w)))->y = (((float2*)(&(__3.w)))->y*((float2*)(&(v__1.w)))->y);
    ((half2*)(&(__1.x)))->x = (half_t)(((float2*)(&(__2.x)))->x);
    ((half2*)(&(__1.x)))->y = (half_t)(((float2*)(&(__2.x)))->y);
    ((half2*)(&(__1.y)))->x = (half_t)(((float2*)(&(__2.y)))->x);
    ((half2*)(&(__1.y)))->y = (half_t)(((float2*)(&(__2.y)))->y);
    ((half2*)(&(__1.z)))->x = (half_t)(((float2*)(&(__2.z)))->x);
    ((half2*)(&(__1.z)))->y = (half_t)(((float2*)(&(__2.z)))->y);
    ((half2*)(&(__1.w)))->x = (half_t)(((float2*)(&(__2.w)))->x);
    ((half2*)(&(__1.w)))->y = (half_t)(((float2*)(&(__2.w)))->y);
    *(uint4*)(((half_t*)buf_dyn_shmem) + (((((i_3 * 1024) + ((((int)threadIdx.x) >> 2) * 32)) + (((((((int)threadIdx.x) & 31) >> 4) + ((((int)threadIdx.x) & 3) >> 1)) & 1) * 16)) + (((((((int)threadIdx.x) & 15) >> 3) + (((int)threadIdx.x) & 1)) & 1) * 8)) + 13312)) = __1;
  }
  tl::cp_async_commit();
  #pragma unroll
  for (int i_4 = 0; i_4 < 2; ++i_4) {
    tl::cp_async_gs<16>(buf_dyn_shmem+(((((i_4 * 2048) + ((((int)threadIdx.x) >> 2) * 64)) + (((((((int)threadIdx.x) & 31) >> 4) + ((((int)threadIdx.x) & 3) >> 1)) & 1) * 32)) + (((((((int)threadIdx.x) & 15) >> 3) + (((int)threadIdx.x) & 1)) & 1) * 16)) + 14336), K+((((((((((int)blockIdx.z) >> 2) * 32768) + (i_4 * 8192)) + ((((int)threadIdx.x) >> 2) * 256)) + ((((int)blockIdx.z) & 3) * 64)) + (((int)blockIdx.y) * 32)) + ((((int)threadIdx.x) & 3) * 8)) + 16384));
  }
  #pragma unroll
  for (int i_5 = 0; i_5 < 2; ++i_5) {
    tl::cp_async_gs<16>(buf_dyn_shmem+(((((i_5 * 2048) + ((((int)threadIdx.x) >> 2) * 64)) + (((((((int)threadIdx.x) & 31) >> 4) + ((((int)threadIdx.x) & 3) >> 1)) & 1) * 32)) + (((((((int)threadIdx.x) & 15) >> 3) + (((int)threadIdx.x) & 1)) & 1) * 16)) + 22528), V+((((((((((int)blockIdx.z) >> 2) * 32768) + (i_5 * 8192)) + ((((int)threadIdx.x) >> 2) * 256)) + ((((int)blockIdx.z) & 3) * 64)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 3) * 8)) + 16384));
  }
  tl::cp_async_commit();
  #pragma unroll
  for (int i_6 = 0; i_6 < 2; ++i_6) {
    uint4 __4;
    ulonglong4 __5;
      ulonglong4 __6;
      uint4 v__2 = *(uint4*)(Q + ((((((((((int)blockIdx.z) >> 2) * 32768) + (i_6 * 8192)) + ((((int)threadIdx.x) >> 2) * 256)) + ((((int)blockIdx.z) & 3) * 64)) + (((int)blockIdx.y) * 32)) + ((((int)threadIdx.x) & 3) * 8)) + 16384));
      ((float2*)(&(__6.x)))->x = (float)(((half2*)(&(v__2.x)))->x);
      ((float2*)(&(__6.x)))->y = (float)(((half2*)(&(v__2.x)))->y);
      ((float2*)(&(__6.y)))->x = (float)(((half2*)(&(v__2.y)))->x);
      ((float2*)(&(__6.y)))->y = (float)(((half2*)(&(v__2.y)))->y);
      ((float2*)(&(__6.z)))->x = (float)(((half2*)(&(v__2.z)))->x);
      ((float2*)(&(__6.z)))->y = (float)(((half2*)(&(v__2.z)))->y);
      ((float2*)(&(__6.w)))->x = (float)(((half2*)(&(v__2.w)))->x);
      ((float2*)(&(__6.w)))->y = (float)(((half2*)(&(v__2.w)))->y);
      ulonglong4 v__3 = make_ulonglong4(*(unsigned long long*)&make_float2(1.250000e-01f, 1.250000e-01f), *(unsigned long long*)&make_float2(1.250000e-01f, 1.250000e-01f), *(unsigned long long*)&make_float2(1.250000e-01f, 1.250000e-01f), *(unsigned long long*)&make_float2(1.250000e-01f, 1.250000e-01f));
      ((float2*)(&(__5.x)))->x = (((float2*)(&(__6.x)))->x*((float2*)(&(v__3.x)))->x);
      ((float2*)(&(__5.x)))->y = (((float2*)(&(__6.x)))->y*((float2*)(&(v__3.x)))->y);
      ((float2*)(&(__5.y)))->x = (((float2*)(&(__6.y)))->x*((float2*)(&(v__3.y)))->x);
      ((float2*)(&(__5.y)))->y = (((float2*)(&(__6.y)))->y*((float2*)(&(v__3.y)))->y);
      ((float2*)(&(__5.z)))->x = (((float2*)(&(__6.z)))->x*((float2*)(&(v__3.z)))->x);
      ((float2*)(&(__5.z)))->y = (((float2*)(&(__6.z)))->y*((float2*)(&(v__3.z)))->y);
      ((float2*)(&(__5.w)))->x = (((float2*)(&(__6.w)))->x*((float2*)(&(v__3.w)))->x);
      ((float2*)(&(__5.w)))->y = (((float2*)(&(__6.w)))->y*((float2*)(&(v__3.w)))->y);
    ((half2*)(&(__4.x)))->x = (half_t)(((float2*)(&(__5.x)))->x);
    ((half2*)(&(__4.x)))->y = (half_t)(((float2*)(&(__5.x)))->y);
    ((half2*)(&(__4.y)))->x = (half_t)(((float2*)(&(__5.y)))->x);
    ((half2*)(&(__4.y)))->y = (half_t)(((float2*)(&(__5.y)))->y);
    ((half2*)(&(__4.z)))->x = (half_t)(((float2*)(&(__5.z)))->x);
    ((half2*)(&(__4.z)))->y = (half_t)(((float2*)(&(__5.z)))->y);
    ((half2*)(&(__4.w)))->x = (half_t)(((float2*)(&(__5.w)))->x);
    ((half2*)(&(__4.w)))->y = (half_t)(((float2*)(&(__5.w)))->y);
    *(uint4*)(((half_t*)buf_dyn_shmem) + (((((i_6 * 1024) + ((((int)threadIdx.x) >> 2) * 32)) + (((((((int)threadIdx.x) & 31) >> 4) + ((((int)threadIdx.x) & 3) >> 1)) & 1) * 16)) + (((((((int)threadIdx.x) & 15) >> 3) + (((int)threadIdx.x) & 1)) & 1) * 8)) + 15360)) = __4;
  }
  tl::cp_async_commit();
  tl::cp_async_wait<2>();
  __syncthreads();
  tl::gemm_ss<64, 64, 32, 2, 2, 0, 1, 1>((&(((half_t*)buf_dyn_shmem)[13312])), (&(((half_t*)buf_dyn_shmem)[5120])), (&(s[0])));
  __syncthreads();
  #pragma unroll
  for (int i_7 = 0; i_7 < 16; ++i_7) {
    for (int vec_s = 0; vec_s < 2; ++vec_s) {
      float condval;
      if (((((((i_7 >> 2) * 16) + ((((int)threadIdx.x) >> 6) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + vec_s) <= ((((((i_7 & 3) >> 1) * 32) + (((((int)threadIdx.x) & 63) >> 5) * 16)) + ((i_7 & 1) * 8)) + ((((int)threadIdx.x) & 31) >> 2)))) {
        condval = s[((i_7 * 2) + vec_s)];
      } else {
        condval = 0.000000e+00f;
      }
      ((half_t*)buf_dyn_shmem)[((((((((((((i_7 & 3) >> 1) * 2048) + (((((int)threadIdx.x) & 63) >> 5) * 1024)) + ((i_7 & 1) * 512)) + (((((int)threadIdx.x) & 31) >> 2) * 64)) + (((((((((i_7 >> 2) * 16) + ((((int)threadIdx.x) >> 6) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + vec_s) >> 5) + ((((int)threadIdx.x) & 31) >> 4)) & 1) * 32)) + (((((((int)threadIdx.x) & 15) >> 3) + ((i_7 & 7) >> 2)) & 1) * 16)) + ((((((int)threadIdx.x) >> 6) + ((((int)threadIdx.x) & 7) >> 2)) & 1) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + vec_s) + 1024)] = ((half_t)condval);
    }
  }
  tl::cp_async_wait<3>();
  __syncthreads();
  tl::gemm_ss<64, 32, 64, 2, 2, 0, 0, 1>((&(((half_t*)buf_dyn_shmem)[1024])), (&(((half_t*)buf_dyn_shmem)[9216])), (&(o[0])));
  __syncthreads();
  #pragma unroll
  for (int i_8 = 0; i_8 < 4; ++i_8) {
    uint1 __7;
    float2 v__4 = *(float2*)(h + (i_8 * 2));
    ((half2*)(&(__7.x)))->x = (half_t)(v__4.x);
    ((half2*)(&(__7.x)))->y = (half_t)(v__4.y);
    *(uint1*)(((half_t*)buf_dyn_shmem) + ((((((((((int)threadIdx.x) & 63) >> 5) * 512) + ((i_8 & 1) * 256)) + (((((int)threadIdx.x) & 31) >> 2) * 32)) + (((((((int)threadIdx.x) & 31) >> 4) + (i_8 >> 1)) & 1) * 16)) + ((((((int)threadIdx.x) >> 6) + ((((int)threadIdx.x) & 15) >> 3)) & 1) * 8)) + ((((int)threadIdx.x) & 3) * 2))) = __7;
  }
  tl::cp_async_wait<3>();
  __syncthreads();
  tl::gemm_ss<32, 32, 64, 2, 2, 1, 0, 0>((&(((half_t*)buf_dyn_shmem)[5120])), (&(((half_t*)buf_dyn_shmem)[9216])), (&(h[0])));
  tl::cp_async_wait<2>();
  __syncthreads();
  tl::gemm_ss<64, 32, 32, 2, 2, 0, 0, 0>((&(((half_t*)buf_dyn_shmem)[13312])), (&(((half_t*)buf_dyn_shmem)[0])), (&(o[0])));
  #pragma unroll
  for (int i_9 = 0; i_9 < 8; ++i_9) {
    uint1 __8;
    float2 v__5 = *(float2*)(o + (i_9 * 2));
    ((half2*)(&(__8.x)))->x = (half_t)(v__5.x);
    ((half2*)(&(__8.x)))->y = (half_t)(v__5.y);
    *(uint1*)(O + (((((((((((((int)blockIdx.y) * 65536) + ((((int)blockIdx.z) >> 2) * 32768)) + (((i_9 & 3) >> 1) * 8192)) + (((((int)threadIdx.x) & 63) >> 5) * 4096)) + ((i_9 & 1) * 2048)) + (((((int)threadIdx.x) & 31) >> 2) * 256)) + ((((int)blockIdx.z) & 3) * 64)) + (((int)blockIdx.x) * 32)) + ((i_9 >> 2) * 16)) + ((((int)threadIdx.x) >> 6) * 8)) + ((((int)threadIdx.x) & 3) * 2))) = __8;
  }
  tl::cp_async_wait<0>();
  __syncthreads();
  tl::gemm_ss<64, 64, 32, 2, 2, 0, 1, 1>((&(((half_t*)buf_dyn_shmem)[15360])), (&(((half_t*)buf_dyn_shmem)[7168])), (&(s[0])));
  __syncthreads();
  #pragma unroll
  for (int i_10 = 0; i_10 < 16; ++i_10) {
    for (int vec_s_1 = 0; vec_s_1 < 2; ++vec_s_1) {
      float condval_1;
      if (((((((i_10 >> 2) * 16) + ((((int)threadIdx.x) >> 6) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + vec_s_1) <= ((((((i_10 & 3) >> 1) * 32) + (((((int)threadIdx.x) & 63) >> 5) * 16)) + ((i_10 & 1) * 8)) + ((((int)threadIdx.x) & 31) >> 2)))) {
        condval_1 = s[((i_10 * 2) + vec_s_1)];
      } else {
        condval_1 = 0.000000e+00f;
      }
      ((half_t*)buf_dyn_shmem)[((((((((((((i_10 & 3) >> 1) * 2048) + (((((int)threadIdx.x) & 63) >> 5) * 1024)) + ((i_10 & 1) * 512)) + (((((int)threadIdx.x) & 31) >> 2) * 64)) + (((((((((i_10 >> 2) * 16) + ((((int)threadIdx.x) >> 6) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + vec_s_1) >> 5) + ((((int)threadIdx.x) & 31) >> 4)) & 1) * 32)) + (((((((int)threadIdx.x) & 15) >> 3) + ((i_10 & 7) >> 2)) & 1) * 16)) + ((((((int)threadIdx.x) >> 6) + ((((int)threadIdx.x) & 7) >> 2)) & 1) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + vec_s_1) + 1024)] = ((half_t)condval_1);
    }
  }
  tl::cp_async_wait<1>();
  __syncthreads();
  tl::gemm_ss<64, 32, 64, 2, 2, 0, 0, 1>((&(((half_t*)buf_dyn_shmem)[1024])), (&(((half_t*)buf_dyn_shmem)[11264])), (&(o[0])));
  __syncthreads();
  #pragma unroll
  for (int i_11 = 0; i_11 < 4; ++i_11) {
    uint1 __9;
    float2 v__6 = *(float2*)(h + (i_11 * 2));
    ((half2*)(&(__9.x)))->x = (half_t)(v__6.x);
    ((half2*)(&(__9.x)))->y = (half_t)(v__6.y);
    *(uint1*)(((half_t*)buf_dyn_shmem) + ((((((((((int)threadIdx.x) & 63) >> 5) * 512) + ((i_11 & 1) * 256)) + (((((int)threadIdx.x) & 31) >> 2) * 32)) + (((((((int)threadIdx.x) & 31) >> 4) + (i_11 >> 1)) & 1) * 16)) + ((((((int)threadIdx.x) >> 6) + ((((int)threadIdx.x) & 15) >> 3)) & 1) * 8)) + ((((int)threadIdx.x) & 3) * 2))) = __9;
  }
  tl::cp_async_wait<1>();
  __syncthreads();
  tl::gemm_ss<32, 32, 64, 2, 2, 1, 0, 0>((&(((half_t*)buf_dyn_shmem)[7168])), (&(((half_t*)buf_dyn_shmem)[11264])), (&(h[0])));
  tl::cp_async_wait<0>();
  __syncthreads();
  tl::gemm_ss<64, 32, 32, 2, 2, 0, 0, 0>((&(((half_t*)buf_dyn_shmem)[15360])), (&(((half_t*)buf_dyn_shmem)[0])), (&(o[0])));
  #pragma unroll
  for (int i_12 = 0; i_12 < 8; ++i_12) {
    uint1 __10;
    float2 v__7 = *(float2*)(o + (i_12 * 2));
    ((half2*)(&(__10.x)))->x = (half_t)(v__7.x);
    ((half2*)(&(__10.x)))->y = (half_t)(v__7.y);
    *(uint1*)(O + ((((((((((((((int)blockIdx.y) * 65536) + ((((int)blockIdx.z) >> 2) * 32768)) + (((i_12 & 3) >> 1) * 8192)) + (((((int)threadIdx.x) & 63) >> 5) * 4096)) + ((i_12 & 1) * 2048)) + (((((int)threadIdx.x) & 31) >> 2) * 256)) + ((((int)blockIdx.z) & 3) * 64)) + (((int)blockIdx.x) * 32)) + ((i_12 >> 2) * 16)) + ((((int)threadIdx.x) >> 6) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 16384)) = __10;
  }
  #pragma unroll
  for (int i_13 = 0; i_13 < 4; ++i_13) {
    *(float2*)(final_state + (((((((((((int)blockIdx.z) * 4096) + (((int)blockIdx.y) * 2048)) + (((((int)threadIdx.x) & 63) >> 5) * 1024)) + ((i_13 & 1) * 512)) + (((((int)threadIdx.x) & 31) >> 2) * 64)) + (((int)blockIdx.x) * 32)) + ((i_13 >> 1) * 16)) + ((((int)threadIdx.x) >> 6) * 8)) + ((((int)threadIdx.x) & 3) * 2))) = *(float2*)(h + (i_13 * 2));
  }
}


#define ERROR_BUF_SIZE 1024
static char error_buf[ERROR_BUF_SIZE];

extern "C" const char* get_last_error() {
    return error_buf;
}

extern "C" int init() {
    error_buf[0] = '\0';
    
    hipError_t result_chunk_linear_attn_fwd_kernel = hipFuncSetAttribute(reinterpret_cast<const void*>(chunk_linear_attn_fwd_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, 34816);
    if (result_chunk_linear_attn_fwd_kernel != hipSuccess) {
        snprintf(error_buf, ERROR_BUF_SIZE, "Failed to set the allowed dynamic shared memory size to %d with error: %s", 34816, hipGetErrorString(result_chunk_linear_attn_fwd_kernel));
        return -1;
    }

    return 0;
}

extern "C" int call(half_t* __restrict__ Q, half_t* __restrict__ K, half_t* __restrict__ V, half_t* __restrict__ O, float* __restrict__ final_state, hipStream_t stream=hipStreamDefault) {
	chunk_linear_attn_fwd_kernel<<<dim3(2, 2, 8), dim3(128, 1, 1), 34816, stream>>>(K, O, Q, V, final_state);
	TILELANG_CHECK_LAST_ERROR("chunk_linear_attn_fwd_kernel");

	return 0;
}
