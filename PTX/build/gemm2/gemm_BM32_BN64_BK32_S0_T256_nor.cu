#include "hip/hip_runtime.h"
#include <tl_templates/cuda/gemm.h>
#include <tl_templates/cuda/copy.h>
#include <tl_templates/cuda/reduce.h>
#include <tl_templates/cuda/ldsm.h>
#include <tl_templates/cuda/threadblock_swizzle.h>
#include <tl_templates/cuda/debug.h>
#ifdef ENABLE_BF16
#include <tl_templates/cuda/cuda_bf16_fallbacks.cuh>
#endif

extern "C" __global__ void main_kernel(half_t* __restrict__ A, half_t* __restrict__ B, half_t* __restrict__ C);
extern "C" __global__ void __launch_bounds__(256, 1) main_kernel(half_t* __restrict__ A, half_t* __restrict__ B, half_t* __restrict__ C) {
  extern __shared__ __align__(1024) uchar buf_dyn_shmem[];
  float C_local[8];
  #pragma unroll
  for (int i = 0; i < 4; ++i) {
    *(float2*)(C_local + (i * 2)) = make_float2(0x0p+0f/*0.000000e+00*/, 0x0p+0f/*0.000000e+00*/);
  }
  for (int k = 0; k < 32; ++k) {
    __syncthreads();
    *(uint2*)(((half_t*)buf_dyn_shmem) + ((((((((int)threadIdx.x) >> 3) * 32) + (((((((int)threadIdx.x) & 63) >> 5) + ((((int)threadIdx.x) & 7) >> 2)) & 1) * 16)) + (((((((int)threadIdx.x) & 31) >> 4) + ((((int)threadIdx.x) & 3) >> 1)) & 1) * 8)) + ((((int)threadIdx.x) & 1) * 4)) + 2048)) = *(uint2*)(A + ((((((int)blockIdx.y) * 32768) + ((((int)threadIdx.x) >> 3) * 1024)) + (k * 32)) + ((((int)threadIdx.x) & 7) * 4)));
    *(uint4*)(((half_t*)buf_dyn_shmem) + ((((((int)threadIdx.x) >> 2) * 32) + (((((((int)threadIdx.x) & 31) >> 4) + ((((int)threadIdx.x) & 3) >> 1)) & 1) * 16)) + (((((((int)threadIdx.x) & 15) >> 3) + (((int)threadIdx.x) & 1)) & 1) * 8))) = *(uint4*)(B + ((((((int)blockIdx.x) * 65536) + ((((int)threadIdx.x) >> 2) * 1024)) + (k * 32)) + ((((int)threadIdx.x) & 3) * 8)));
    __syncthreads();
    tl::gemm_ss<32, 64, 32, 2, 4, 0, 1, 0, 32, 32, 0, 0>((&(((half_t*)buf_dyn_shmem)[2048])), (&(((half_t*)buf_dyn_shmem)[0])), (&(C_local[0])));
  }
  #pragma unroll
  for (int i_1 = 0; i_1 < 4; ++i_1) {
    uint1 __1;
    float2 v_ = *(float2*)(C_local + (i_1 * 2));
    ((half2*)(&(__1.x)))->x = (half_t)(v_.x);
    ((half2*)(&(__1.x)))->y = (half_t)(v_.y);
    *(uint1*)(C + ((((((((((int)blockIdx.y) * 32768) + (((((int)threadIdx.x) & 63) >> 5) * 16384)) + ((i_1 & 1) * 8192)) + (((((int)threadIdx.x) & 31) >> 2) * 1024)) + (((int)blockIdx.x) * 64)) + ((i_1 >> 1) * 32)) + ((((int)threadIdx.x) >> 6) * 8)) + ((((int)threadIdx.x) & 3) * 2))) = __1;
  }
}


#define ERROR_BUF_SIZE 1024
static char error_buf[ERROR_BUF_SIZE];

extern "C" const char* get_last_error() {
    return error_buf;
}

extern "C" int init() {
    error_buf[0] = '\0';
    
    hipError_t result_main_kernel = hipFuncSetAttribute(reinterpret_cast<const void*>(main_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, 6144);
    if (result_main_kernel != hipSuccess) {
        snprintf(error_buf, ERROR_BUF_SIZE, "Failed to set the allowed dynamic shared memory size to %d with error: %s", 6144, hipGetErrorString(result_main_kernel));
        return -1;
    }

    return 0;
}

extern "C" int call(half_t* __restrict__ A, half_t* __restrict__ B, half_t* __restrict__ C, hipStream_t stream=hipStreamDefault) {
	main_kernel<<<dim3(16, 32, 1), dim3(256, 1, 1), 6144, stream>>>(A, B, C);
	TILELANG_CHECK_LAST_ERROR("main_kernel");

	return 0;
}
